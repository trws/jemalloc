#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
__global__ void add_one(int *a) {
    *a += 1;
}

void test(int * a) {
    add_one<<<1,1>>>(a);
    hipDeviceSynchronize();

    printf("%d\n", *a);
}

int main(int argc, char *argv[])
{
    int *a2 = (int*)calloc(1, sizeof(int));
    test(a2);

    if (*a2 == 1)
        printf("success!\n");
    else
        printf("failure!\n");



    return 0;
}
